#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "cuda_kernels.h"

void cu_select_insert_internal( CudaImg &img_big, CudaImg &img_small, int2 pos, bool select )
{
    hipError_t cudaErr;

    int block_size = 16;
    dim3 block_count;
    block_count.x = img_small.m_size.x / block_size + (img_small.m_size.x % block_size ? 1 : 0);
    block_count.y = img_small.m_size.y / block_size + (img_small.m_size.y % block_size ? 1 : 0);

    kernel_select_insert<<<block_count, dim3(block_size, block_size)>>>(img_big, img_small, pos, select);

    if ((cudaErr = hipGetLastError()) != hipSuccess)
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cudaErr ) );
}

void cu_rotate_90_internal( CudaImg &img_orig, CudaImg &img_rotated, bool clockwise )
{
    hipError_t cudaErr;

    int block_size = 16;
    dim3 block_count;
    block_count.x = img_orig.m_size.x / block_size + (img_orig.m_size.x % block_size ? 1 : 0);
    block_count.y = img_orig.m_size.y / block_size + (img_orig.m_size.y % block_size ? 1 : 0);

    kernel_rotate_90<<<block_count, dim3(block_size, block_size)>>>(img_orig, img_rotated, clockwise);

    if ((cudaErr = hipGetLastError()) != hipSuccess)
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cudaErr ) );
}

void cu_decrease_res(CudaImg &img_orig, CudaImg &img_small)
{
    CudaImg img_orig_internal = img_orig.m_size;
    CudaImg img_small_internal = img_small.m_size;

    cudaErrCheck(hipMalloc(&img_orig_internal.m_p_void, img_orig.m_size.x * img_orig.m_size.y * 4));
    cudaErrCheck(hipMemcpy(img_orig_internal.m_p_void, img_orig.m_p_void, img_orig.m_size.x * img_orig.m_size.y * 4, hipMemcpyHostToDevice));

    cudaErrCheck(hipMalloc(&img_small_internal.m_p_void, img_small_internal.m_size.x * img_small_internal.m_size.y * 4));
    

    cu_decrease_res_internal(img_orig_internal, img_small_internal);


    cudaErrCheck(hipMemcpy(img_small.m_p_void, img_small_internal.m_p_void, img_small.m_size.x * img_small.m_size.y * 4, hipMemcpyDeviceToHost));

    cudaErrCheck(hipFree(img_orig_internal.m_p_void));
    cudaErrCheck(hipFree(img_small_internal.m_p_void));
}

void cu_decrease_res_internal( CudaImg &img_orig, CudaImg &img_small )
{
	hipError_t cudaErr;

	int block_size = 16;
	dim3 block_count;
	block_count.x = img_small.m_size.x / block_size + (img_small.m_size.x % block_size ? 1 : 0);
	block_count.y = img_small.m_size.y / block_size + (img_small.m_size.y % block_size ? 1 : 0);

	kernel_decrease_res<<<block_count, dim3(block_size, block_size)>>>(img_orig, img_small);

	if ((cudaErr = hipGetLastError()) != hipSuccess)
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cudaErr ) );
}

void cu_split(CudaImg &img_orig, CudaImg &img_upper, CudaImg &img_lower)
{
	CudaImg img_orig_int(img_orig.m_size);
	CudaImg img_upper_int(img_upper.m_size);
	CudaImg img_lower_int(img_lower.m_size);

	cudaErrCheck(hipMalloc(&img_orig_int.m_p_void, img_orig_int.m_size.x * img_orig_int.m_size.y * 4));
	cudaErrCheck(hipMemcpy(img_orig_int.m_p_void, img_orig.m_p_void, img_orig_int.m_size.x * img_orig_int.m_size.y * 4, hipMemcpyHostToDevice));

	cudaErrCheck(hipMalloc(&img_upper_int.m_p_void, img_upper_int.m_size.x * img_upper_int.m_size.y * 4));
	cudaErrCheck(hipMalloc(&img_lower_int.m_p_void, img_lower_int.m_size.x * img_lower_int.m_size.y * 4));

	cu_split_internal(img_orig_int, img_upper_int, img_lower_int);

	cudaErrCheck(hipMemcpy(img_upper.m_p_void, img_upper_int.m_p_void, img_upper.m_size.x * img_upper.m_size.y * 4, hipMemcpyDeviceToHost));
	cudaErrCheck(hipMemcpy(img_lower.m_p_void, img_lower_int.m_p_void, img_lower.m_size.x * img_lower.m_size.y * 4, hipMemcpyDeviceToHost));

    cudaErrCheck(hipFree(img_orig_int.m_p_void));
    cudaErrCheck(hipFree(img_upper_int.m_p_void));
    cudaErrCheck(hipFree(img_lower_int.m_p_void));
}

void cu_split_internal(CudaImg &img_orig, CudaImg &img_upper, CudaImg &img_lower)
{
	hipError_t cudaErr;

	int block_size = 16;
	dim3 block_count;
	block_count.x = img_upper.m_size.x / block_size + (img_upper.m_size.x % block_size ? 1 : 0);
	block_count.y = img_upper.m_size.y / block_size + (img_upper.m_size.y % block_size ? 1 : 0);

	kernel_select_insert<<<block_count, dim3(block_size, block_size)>>>(img_orig, img_upper, {0, 0}, true);
	kernel_select_insert<<<block_count, dim3(block_size, block_size)>>>(img_orig, img_lower, {0, (int)img_upper.m_size.y}, true);

	if ((cudaErr = hipGetLastError()) != hipSuccess)
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cudaErr ) );
}

void cu_clear_internal(CudaImg &img)
{
	hipError_t cudaErr;

	int block_size = 16;
	dim3 block_count;
	block_count.x = img.m_size.x / block_size + (img.m_size.x % block_size ? 1 : 0);
	block_count.y = img.m_size.y / block_size + (img.m_size.y % block_size ? 1 : 0);

	kernel_clear<<<block_count, dim3(block_size, block_size)>>>(img);

	if ((cudaErr = hipGetLastError()) != hipSuccess)
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cudaErr ) );
}
