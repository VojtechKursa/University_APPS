#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void kernel_add_arrays(int* arrIn1, int* arrIn2, int* arrOut, int length)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= length) return;

    arrOut[i] = arrIn1[i] + arrIn2[i];
}


void cu_add_arrays(int* arrIn1, int* arrIn2, int* arrOut, int length)
{
    int threadsPerBlock = 128;
    int blockCount = length / threadsPerBlock + (length % threadsPerBlock ? 1 : 0);

    int lengthInBytes = sizeof(int) * length;
    int *arrIn1GPU, *arrIn2GPU, *arrOutGPU;

    hipMalloc(&arrIn1GPU, lengthInBytes);
    hipMalloc(&arrIn2GPU, lengthInBytes);
    hipMalloc(&arrOutGPU, lengthInBytes);

    hipMemcpy(arrIn1GPU, arrIn1, lengthInBytes, hipMemcpyHostToDevice);
    hipMemcpy(arrIn2GPU, arrIn2, lengthInBytes, hipMemcpyHostToDevice);

    kernel_add_arrays<<<blockCount, threadsPerBlock>>>(arrIn1GPU, arrIn2GPU, arrOutGPU, length);

    hipMemcpy(arrOut, arrOutGPU, lengthInBytes, hipMemcpyDeviceToHost);

    hipFree(arrIn1GPU);
    hipFree(arrIn2GPU);
    hipFree(arrOutGPU);
}